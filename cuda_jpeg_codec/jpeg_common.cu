#include "hip/hip_runtime.h"
﻿#include "jpeg_common.cuh"

/*--------------------标准编码表--------------------*/
//详情见enum StaticTable
static unsigned char _ZIGZAG[64] = {
		0,  1,  5,  6, 14, 15, 27, 28,
		2,  4,  7, 13, 16, 26, 29, 42,
		3,  8, 12, 17, 25, 30, 41, 43,
		9, 11, 18, 24, 31, 40, 44, 53,
		10, 19, 23, 32, 39, 45, 52, 54,
		20, 22, 33, 38, 46, 51, 55, 60,
		21, 34, 37, 47, 50, 56, 59, 61,
		35, 36, 48, 49, 57, 58, 62, 63
};

/*APP0 应用程序标记段
2 Bytes | 标记码 0xFFE0
2 Bytes | 数据段长度，包含本字段，但不包括标记码
5 Bytes | 标识符 0x4A46494600，JFIF0的ASCII表示(固定值)
1 Bytes | 主版本号,0x01(版本号表示JFIF的版本1.2)
1 Bytes | 副版本号,0x02
1 Bytes | 图像密度单位0x00(0:无单位,1:点数/英寸,2:点数/厘米)
2 Bytes | X方向像素密度0x00,0x01
2 Bytes | Y方向像素密度0x00,0x01
1 Bytes | 缩略图水平像素数目0x00
1 Bytes | 缩略图垂直像素数目0x00
n Bytes | 缩略图，RGB24位图数据,当上面俩个为0x00时表示无.
*/
static unsigned char _APP0_TAG[14] =
{
	0x4a, 0x46, 0x49, 0x46, 0x00,
	0x01, 0x02,
	0x00,
	0x00, 0x01, 0x00, 0x01,
	0x00, 0x00
};
unsigned char _STD_Y_QT[64] =
{
	16, 11, 10, 16, 24, 40, 51, 61,
	12, 12, 14, 19, 26, 58, 60, 55,
	14, 13, 16, 24, 40, 57, 69, 56,
	14, 17, 22, 29, 51, 87, 80, 62,
	18, 22, 37, 56, 68, 109, 103, 77,
	24, 35, 55, 64, 81, 104, 113, 92,
	49, 64, 78, 87, 103, 121, 120, 101,
	72, 92, 95, 98, 112, 100, 103, 99
};
unsigned char _STD_UV_QT[64] =
{
	17, 18, 24, 47, 99, 99, 99, 99,
	18, 21, 26, 66, 99, 99, 99, 99,
	24, 26, 56, 99, 99, 99, 99, 99,
	47, 66, 99, 99, 99, 99, 99, 99,
	99, 99, 99, 99, 99, 99, 99, 99,
	99, 99, 99, 99, 99, 99, 99, 99,
	99, 99, 99, 99, 99, 99, 99, 99,
	99, 99, 99, 99, 99, 99, 99, 99
};
static unsigned char _STD_DC_Y_NRCODES[16] = { 0, 1, 5, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0 };
static unsigned char _STD_DC_Y_VALUES[12] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11 };
static unsigned char _STD_DC_UV_NRCODES[16] = { 0, 3, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0 };
static unsigned char _STD_DC_UV_VALUES[12] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11 };
static unsigned char _STD_AC_Y_NRCODES[16] = { 0, 2, 1, 3, 3, 2, 4, 3, 5, 5, 4, 4, 0, 0, 1, 0X7D };
static unsigned char _STD_AC_Y_VALUES[162] =
{
	0x01, 0x02, 0x03, 0x00, 0x04, 0x11, 0x05, 0x12,
	0x21, 0x31, 0x41, 0x06, 0x13, 0x51, 0x61, 0x07,
	0x22, 0x71, 0x14, 0x32, 0x81, 0x91, 0xa1, 0x08,
	0x23, 0x42, 0xb1, 0xc1, 0x15, 0x52, 0xd1, 0xf0,
	0x24, 0x33, 0x62, 0x72, 0x82, 0x09, 0x0a, 0x16,
	0x17, 0x18, 0x19, 0x1a, 0x25, 0x26, 0x27, 0x28,
	0x29, 0x2a, 0x34, 0x35, 0x36, 0x37, 0x38, 0x39,
	0x3a, 0x43, 0x44, 0x45, 0x46, 0x47, 0x48, 0x49,
	0x4a, 0x53, 0x54, 0x55, 0x56, 0x57, 0x58, 0x59,
	0x5a, 0x63, 0x64, 0x65, 0x66, 0x67, 0x68, 0x69,
	0x6a, 0x73, 0x74, 0x75, 0x76, 0x77, 0x78, 0x79,
	0x7a, 0x83, 0x84, 0x85, 0x86, 0x87, 0x88, 0x89,
	0x8a, 0x92, 0x93, 0x94, 0x95, 0x96, 0x97, 0x98,
	0x99, 0x9a, 0xa2, 0xa3, 0xa4, 0xa5, 0xa6, 0xa7,
	0xa8, 0xa9, 0xaa, 0xb2, 0xb3, 0xb4, 0xb5, 0xb6,
	0xb7, 0xb8, 0xb9, 0xba, 0xc2, 0xc3, 0xc4, 0xc5,
	0xc6, 0xc7, 0xc8, 0xc9, 0xca, 0xd2, 0xd3, 0xd4,
	0xd5, 0xd6, 0xd7, 0xd8, 0xd9, 0xda, 0xe1, 0xe2,
	0xe3, 0xe4, 0xe5, 0xe6, 0xe7, 0xe8, 0xe9, 0xea,
	0xf1, 0xf2, 0xf3, 0xf4, 0xf5, 0xf6, 0xf7, 0xf8,
	0xf9, 0xfa
};
static unsigned char _STD_AC_UV_NRCODES[16] = { 0, 2, 1, 2, 4, 4, 3, 4, 7, 5, 4, 4, 0, 1, 2, 0X77 };
static unsigned char _STD_AC_UV_VALUES[162] =
{
	0x00, 0x01, 0x02, 0x03, 0x11, 0x04, 0x05, 0x21,
	0x31, 0x06, 0x12, 0x41, 0x51, 0x07, 0x61, 0x71,
	0x13, 0x22, 0x32, 0x81, 0x08, 0x14, 0x42, 0x91,
	0xa1, 0xb1, 0xc1, 0x09, 0x23, 0x33, 0x52, 0xf0,
	0x15, 0x62, 0x72, 0xd1, 0x0a, 0x16, 0x24, 0x34,
	0xe1, 0x25, 0xf1, 0x17, 0x18, 0x19, 0x1a, 0x26,
	0x27, 0x28, 0x29, 0x2a, 0x35, 0x36, 0x37, 0x38,
	0x39, 0x3a, 0x43, 0x44, 0x45, 0x46, 0x47, 0x48,
	0x49, 0x4a, 0x53, 0x54, 0x55, 0x56, 0x57, 0x58,
	0x59, 0x5a, 0x63, 0x64, 0x65, 0x66, 0x67, 0x68,
	0x69, 0x6a, 0x73, 0x74, 0x75, 0x76, 0x77, 0x78,
	0x79, 0x7a, 0x82, 0x83, 0x84, 0x85, 0x86, 0x87,
	0x88, 0x89, 0x8a, 0x92, 0x93, 0x94, 0x95, 0x96,
	0x97, 0x98, 0x99, 0x9a, 0xa2, 0xa3, 0xa4, 0xa5,
	0xa6, 0xa7, 0xa8, 0xa9, 0xaa, 0xb2, 0xb3, 0xb4,
	0xb5, 0xb6, 0xb7, 0xb8, 0xb9, 0xba, 0xc2, 0xc3,
	0xc4, 0xc5, 0xc6, 0xc7, 0xc8, 0xc9, 0xca, 0xd2,
	0xd3, 0xd4, 0xd5, 0xd6, 0xd7, 0xd8, 0xd9, 0xda,
	0xe2, 0xe3, 0xe4, 0xe5, 0xe6, 0xe7, 0xe8, 0xe9,
	0xea, 0xf2, 0xf3, 0xf4, 0xf5, 0xf6, 0xf7, 0xf8,
	0xf9, 0xfa
};

namespace jpeg_common {

	namespace device {

	}

	void jpeg_common::WriteJpeg(std::string &output_fname, unsigned char *pDstJpeg,int &nOutputLenth)
	{
		//写结果到文件
		std::ofstream outputFile(output_fname, std::ios::binary|std::ios::out);
		outputFile.write(reinterpret_cast<const char *>(pDstJpeg), nOutputLenth);
	}

	//TODO 使用自定义量化函数生成量化表,更方便调整图像质量
	unsigned char * jpeg_common::getStaticTable(int tableName)
	{
		switch (tableName)
		{
		case (StaticTable::ZIGZAG):
			return _ZIGZAG;
		case (StaticTable::APP0_TAG):
			return _APP0_TAG;
		case (StaticTable::STD_Y_QT):
			return _STD_Y_QT;
		case (StaticTable::STD_UV_QT):
			return _STD_UV_QT;
		case (StaticTable::STD_DC_Y_NRCODES):
			return _STD_DC_Y_NRCODES;
		case (StaticTable::STD_DC_Y_VALUES):
			return _STD_DC_Y_VALUES;
		case (StaticTable::STD_DC_UV_NRCODES):
			return _STD_DC_UV_NRCODES;
		case (StaticTable::STD_DC_UV_VALUES):
			return _STD_DC_UV_VALUES;
		case (StaticTable::STD_AC_Y_NRCODES):
			return _STD_AC_Y_NRCODES;
		case (StaticTable::STD_AC_Y_VALUES):
			return _STD_AC_Y_VALUES;
		case (StaticTable::STD_AC_UV_NRCODES):
			return _STD_AC_UV_NRCODES;
		case (StaticTable::STD_AC_UV_VALUES):
			return _STD_AC_UV_VALUES;
		default:
			std::cerr << "未识别的静态表" << std::endl;
			return NULL;
		}
	}

	void jpeg_common::setQTByQuality(int nQualityFactor) {
		nppiQuantFwdRawTableInit_JPEG_8u(_STD_Y_QT, nQualityFactor);
		nppiQuantFwdRawTableInit_JPEG_8u(_STD_UV_QT, nQualityFactor);
		/*与npp实现等价
		float s;
		if (nQualityFactor < 50)
			s = 5000.0f / nQualityFactor;
		else s = 200.0f - 2 * nQualityFactor;

		for (size_t i = 0; i < 64; i++) {
			// 亮度
			float luminVal = (float)_STD_Y_QT[i];
			luminVal = floor((s * luminVal + 50.0f) / 100.0f);
			if (luminVal < 1)
				luminVal = 1;
			else if (luminVal > 255)
				luminVal = 255;
			_STD_Y_QT[i] = (unsigned char)luminVal;
			// 色度
			float chromaVal = (float)_STD_UV_QT[i];
			chromaVal = floor((s * chromaVal + 50.0f) / 100.0f);
			if (chromaVal < 1)
				chromaVal = 1;
			else if (chromaVal > 255)
				chromaVal = 255;
			_STD_UV_QT[i] = (unsigned char)chromaVal;
		}
		*/
	}

	int getStaticTableLenth(int tableName)
	{
		switch (tableName)
		{
		case (StaticTable::ZIGZAG):
			return sizeof(_ZIGZAG);
		case (StaticTable::APP0_TAG):
			return sizeof(_APP0_TAG);
		case (StaticTable::STD_Y_QT):
			return sizeof(_STD_Y_QT);
		case (StaticTable::STD_UV_QT):
			return sizeof(_STD_UV_QT);
		case (StaticTable::STD_DC_Y_NRCODES):
			return sizeof(_STD_DC_Y_NRCODES);
		case (StaticTable::STD_DC_Y_VALUES):
			return sizeof(_STD_DC_Y_VALUES);
		case (StaticTable::STD_DC_UV_NRCODES):
			return sizeof(_STD_DC_UV_NRCODES);
		case (StaticTable::STD_DC_UV_VALUES):
			return sizeof(_STD_DC_UV_VALUES);
		case (StaticTable::STD_AC_Y_NRCODES):
			return sizeof(_STD_AC_Y_NRCODES);
		case (StaticTable::STD_AC_Y_VALUES):
			return sizeof(_STD_AC_Y_VALUES);
		case (StaticTable::STD_AC_UV_NRCODES):
			return sizeof(_STD_AC_UV_NRCODES);
		case (StaticTable::STD_AC_UV_VALUES):
			return sizeof(_STD_AC_UV_VALUES);
		default:
			std::cerr << "未识别的静态表" << std::endl;
			return -1;
		}
	}

	int getStaticTableSize(int tableName)
	{
		switch (tableName)
		{
		case (StaticTable::ZIGZAG):
			return sizeof(_ZIGZAG) / sizeof(unsigned char);
		case (StaticTable::APP0_TAG):
			return sizeof(_APP0_TAG) / sizeof(unsigned char);
		case (StaticTable::STD_Y_QT):
			return sizeof(_STD_Y_QT) / sizeof(unsigned char);
		case (StaticTable::STD_UV_QT):
			return sizeof(_STD_UV_QT) / sizeof(unsigned char);
		case (StaticTable::STD_DC_Y_NRCODES):
			return sizeof(_STD_DC_Y_NRCODES) / sizeof(unsigned char);
		case (StaticTable::STD_DC_Y_VALUES):
			return sizeof(_STD_DC_Y_VALUES) / sizeof(unsigned char);
		case (StaticTable::STD_DC_UV_NRCODES):
			return sizeof(_STD_DC_UV_NRCODES) / sizeof(unsigned char);
		case (StaticTable::STD_DC_UV_VALUES):
			return sizeof(_STD_DC_UV_VALUES) / sizeof(unsigned char);
		case (StaticTable::STD_AC_Y_NRCODES):
			return sizeof(_STD_AC_Y_NRCODES) / sizeof(unsigned char);
		case (StaticTable::STD_AC_Y_VALUES):
			return sizeof(_STD_AC_Y_VALUES) / sizeof(unsigned char);
		case (StaticTable::STD_AC_UV_NRCODES):
			return sizeof(_STD_AC_UV_NRCODES) / sizeof(unsigned char);
		case (StaticTable::STD_AC_UV_VALUES):
			return sizeof(_STD_AC_UV_VALUES) / sizeof(unsigned char);
		default:
			std::cerr << "未识别的静态表" << std::endl;
			return -1;
		}
	}

	bool jpeg_common::isValidJpeg(unsigned char * pJpegData, int &nPos, int nInputLength)
	{
		int nMarker = nextMarker(pJpegData, nPos, nInputLength);
		return nMarker == SOI ? true : false;

	}

	template<class T>
	T readBigEndian(const unsigned char *pData)
	{
		if (sizeof(T) > 1)
		{
			unsigned char p[sizeof(T)];
			//反转 [first, last) 范围中的元素顺序,并复制到dst
			//JPEG文件格式中，一个字(16位)的存储使用的是Motorola格式，而不是Intel格式.
			//也就是说，一个字的高字节(高8位)在数据流的前面，低字节(低8位)在数据流的后面
			std::reverse_copy(pData, pData + sizeof(T), p);
			return *reinterpret_cast<T *>(p);
		}
		else
		{
			return *pData;
		}
	}

	template<class T>
	void writeBigEndian(unsigned char *pData, T value)
	{
		unsigned char *pValue = reinterpret_cast<unsigned char *>(&value);
		//反转 [first, last) 范围中的元素顺序,并复制到dst
		//JPEG文件格式中，一个字(16位)的存储使用的是Motorola格式，而不是Intel格式.
		//也就是说，一个字的高字节(高8位)在数据流的前面，低字节(低8位)在数据流的后面
		std::reverse_copy(pValue, pValue + sizeof(T), pData);
	}

	int jpeg_common::DivUp(int x, int d)
	{
		if (d != 0) {
			return (x + d - 1) / d;
		}
		else {
			return -1;
		}

	}

	template<typename T>
	T readAndAdvance(const unsigned char *&pData)
	{
		T nElement = readBigEndian<T>(pData);
		pData += sizeof(T);
		return nElement;
	}

	template<typename T>
	void writeAndAdvance(unsigned char *&pData, T nElement)
	{
		writeBigEndian<T>(pData, nElement);
		pData += sizeof(T);
	}

	int jpeg_common::nextMarker(const unsigned char *pData, int &nPos, int nLength)
	{
		unsigned char c = pData[nPos++];

		do
		{
			//0xff=255 u为无符号的意思
			while (c != 0xFFU && nPos < nLength)
			{
				c = pData[nPos++];
			}

			if (nPos >= nLength)
				return -1;

			//c == 0xffu
			c = pData[nPos++];
		} while (c == 0 || c == 0x0FFU);

		return c;
	}

	void jpeg_common::writeMarker(unsigned char nMarker, unsigned char *&pData)
	{
		*pData++ = FF;//写入标记码的标记,指针++
		*pData++ = nMarker;//补充标记码信息,指针++
	}

	void jpeg_common::writeJFIFTag(unsigned char *&pData)
	{
		writeMarker(APP0, pData);//APP0,Application保留标记
		writeAndAdvance<unsigned short>(pData, sizeof(APP0_TAG) + 2);//写入数据段长度,包含本身2字节,但不包括标记码
		memcpy(pData, getStaticTable(StaticTable::APP0_TAG), getStaticTableLenth(StaticTable::APP0_TAG));//写入标记段数据
		pData += sizeof(APP0_TAG);
	}

	void jpeg_common::readFrameHeader(const unsigned char *pData, FrameHeader &header)
	{
		readAndAdvance<unsigned short>(pData);
		header.nSamplePrecision = readAndAdvance<unsigned char>(pData);
		header.nHeight = readAndAdvance<unsigned short>(pData);
		header.nWidth = readAndAdvance<unsigned short>(pData);
		header.nComponents = readAndAdvance<unsigned char>(pData);

		for (int i = 0; i < header.nComponents; ++i)
		{
			header.aComponentIdentifier[i] = readAndAdvance<unsigned char>(pData);
			header.aSamplingFactors[i] = readAndAdvance<unsigned char>(pData);
			header.aQuantizationTableSelector[i] = readAndAdvance<unsigned char>(pData);
		}

	}

	void jpeg_common::writeFrameHeader(const FrameHeader &header, unsigned char *&pData)
	{
		unsigned char aTemp[128];
		unsigned char *pTemp = aTemp;

		writeAndAdvance<unsigned char>(pTemp, header.nSamplePrecision);
		writeAndAdvance<unsigned short>(pTemp, header.nHeight);
		writeAndAdvance<unsigned short>(pTemp, header.nWidth);
		writeAndAdvance<unsigned char>(pTemp, header.nComponents);

		for (int i = 0; i < header.nComponents; ++i)
		{
			writeAndAdvance<unsigned char>(pTemp, header.aComponentIdentifier[i]);
			writeAndAdvance<unsigned char>(pTemp, header.aSamplingFactors[i]);
			writeAndAdvance<unsigned char>(pTemp, header.aQuantizationTableSelector[i]);
		}

		unsigned short nLength = (unsigned short)(pTemp - aTemp);//获取长度

		writeMarker(SOF0, pData);//SOF0,图像帧开始
		writeAndAdvance<unsigned short>(pData, nLength + 2); //写入数据段长度,包含本身2字节,但不包括标记码
		memcpy(pData, aTemp, nLength);//写入标记段数据
		pData += nLength;
	}

	/*TODO 只考虑到了读取到了正确头结构时的处理，
	如果读取到了错误的信息，就会导致segmentfault，
	由于例程只支持彩色jpeg图片的处理，
	所以在读取到扫描头的nComponents字段时判断是否为3，如果不是则说明头部错误。
	*/
	void jpeg_common::readScanHeader(const unsigned char *pData, ScanHeader &header)
	{
		readAndAdvance<unsigned short>(pData);

		header.nComponents = readAndAdvance<unsigned char>(pData);

		for (int i = 0; i < header.nComponents; ++i)
		{
			header.aComponentSelector[i] = readAndAdvance<unsigned char>(pData);
			header.aHuffmanTablesSelector[i] = readAndAdvance<unsigned char>(pData);
		}

		header.nSs = readAndAdvance<unsigned char>(pData);
		header.nSe = readAndAdvance<unsigned char>(pData);
		header.nA = readAndAdvance<unsigned char>(pData);
	}

	void jpeg_common::writeScanHeader(const ScanHeader &header, unsigned char *&pData)
	{
		unsigned char aTemp[128];
		unsigned char *pTemp = aTemp;

		writeAndAdvance<unsigned char>(pTemp, header.nComponents);

		for (int c = 0; c < header.nComponents; ++c)
		{
			writeAndAdvance<unsigned char>(pTemp, header.aComponentSelector[c]);
			writeAndAdvance<unsigned char>(pTemp, header.aHuffmanTablesSelector[c]);
		}

		writeAndAdvance<unsigned char>(pTemp, header.nSs);
		writeAndAdvance<unsigned char>(pTemp, header.nSe);
		writeAndAdvance<unsigned char>(pTemp, header.nA);

		unsigned short nLength = (unsigned short)(pTemp - aTemp);

		writeMarker(SOS, pData);//SOS,Start of Scan,扫描开始
		writeAndAdvance<unsigned short>(pData, nLength + 2);//写入数据段长度,包含本身2字节,但不包括标记码
		memcpy(pData, aTemp, nLength);//写入标记段数据
		pData += nLength;
	}

	void jpeg_common::readQuantizationTables(const unsigned char *pData, QuantizationTable *pTables)
	{
		unsigned short nLength = readAndAdvance<unsigned short>(pData) - 2;

		while (nLength > 0)
		{
			unsigned char nPrecisionAndIdentifier = readAndAdvance<unsigned char>(pData);

			//按位取与,保留低位(ID),高位清0
			int nIdentifier = nPrecisionAndIdentifier & 0x0f;

			pTables[nIdentifier].nPrecisionAndIdentifier = nPrecisionAndIdentifier;
			memcpy(pTables[nIdentifier].aTable, pData, 64);//写入标记段数据
			pData += 64;
			nLength -= 65;
		}
	}

	void jpeg_common::writeQuantizationTable(const QuantizationTable &table, unsigned char *&pData)
	{
		writeMarker(DQT, pData);//DQT,Define Quantization Table,定义量化表
		writeAndAdvance<unsigned short>(pData, sizeof(QuantizationTable) + 2);//写入数据段长度,包含本身2字节,但不包括标记码
		memcpy(pData, &table, sizeof(QuantizationTable));//写入标记段数据
		pData += sizeof(QuantizationTable);
	}

	void jpeg_common::writeHuffmanTable(const HuffmanTable &table, unsigned char *&pData)
	{
		writeMarker(DHT, pData);//DHT,Difine Huffman Table,定义哈夫曼表

		int nCodeCount = 0;

		for (int i = 0; i < 16; ++i)
		{
			nCodeCount += table.aCodes[i];
		}

		writeAndAdvance<unsigned short>(pData, 17 + nCodeCount + 2);//段长度2字节+HT信息1字节+HT位表16字节+HT值表256字节
		memcpy(pData, &table, 17 + nCodeCount);//写入标记段
		pData += 17 + nCodeCount;
	}
	/*TODO 只考虑了正确的情况，要分别判断nClass变量，nIdx变量和nCodeCount变量的值是否合法。
	*/
	void jpeg_common::readHuffmanTables(const unsigned char *pData, HuffmanTable *pTables)
	{
		unsigned short nLength = readAndAdvance<unsigned short>(pData) - 2;//段长度-2为段内容长度

		while (nLength > 0)
		{
			unsigned char nClassAndIdentifier = readAndAdvance<unsigned char>(pData);
			//获取高位类型信息AC or DC
			int nClass = nClassAndIdentifier >> 4;
			//按位取与,保留低位(ID),高位清0
			int nIdentifier = nClassAndIdentifier & 0x0f;
			int nIdx = nClass * 2 + nIdentifier;//00亮度DC表,10亮度AC表，01色度DC表,11色度AC表
			pTables[nIdx].nClassAndIdentifier = nClassAndIdentifier;

			int nCodeCount = 0;

			for (int i = 0; i < 16; ++i)
			{
				pTables[nIdx].aCodes[i] = readAndAdvance<unsigned char>(pData);
				nCodeCount += pTables[nIdx].aCodes[i];
			}

			memcpy(pTables[nIdx].aTable, pData, nCodeCount);
			pData += nCodeCount;

			nLength -= (17 + nCodeCount);
		}
	}

	void jpeg_common::readRestartInterval(const unsigned char *pData, int &nRestartInterval)
	{
		readAndAdvance<unsigned short>(pData);
		nRestartInterval = readAndAdvance<unsigned short>(pData);
	}


}