#include "hip/hip_runtime.h"
﻿#include "jpeg_encoder.cuh"

using namespace std;
using namespace jpeg_common;

CudaJpegEncoder::CudaJpegEncoder()
{
}

CudaJpegEncoder::CudaJpegEncoder(int width, int height, int channel, int quality)
{
	nRestartInterval = -1;//解码及单通道编码时复位间隔参数

	/*--------------------分配DCT状态结构显存--------------------*/
	NPP_CHECK_NPP(nppiDCTInitAlloc(&pDCTState));

	/*--------------------帧头初始化--------------------*/
	memset(&oFrameHeader, 0, sizeof(FrameHeader));
	oFrameHeader.nSamplePrecision = 8;
	oFrameHeader.nComponents = static_cast<unsigned char>(channel);
	oFrameHeader.aComponentIdentifier[0] = 1;//Y
	oFrameHeader.aComponentIdentifier[1] = 2;//Cb
	oFrameHeader.aComponentIdentifier[2] = 3;//Cr
	oFrameHeader.nWidth = static_cast<unsigned short>(width);
	oFrameHeader.nHeight = static_cast<unsigned short>(height);

	if (channel == 1)
	{
		//TODO 支持直接的单通道处理,单通道需要检测是否正确初始化
		// oFrameHeader.aSamplingFactors[0] = 1;
		// oFrameHeader.aQuantizationTableSelector[0] = 0;

		oFrameHeader.aSamplingFactors[0] = 34;
		oFrameHeader.aSamplingFactors[1] = 17;
		oFrameHeader.aSamplingFactors[2] = 17;
		oFrameHeader.aQuantizationTableSelector[0] = 0;
		oFrameHeader.aQuantizationTableSelector[1] = 1;
		oFrameHeader.aQuantizationTableSelector[2] = 1;
	}
	else if (channel == 3)
	{
		oFrameHeader.aSamplingFactors[0] = 34;
		oFrameHeader.aSamplingFactors[1] = 17;
		oFrameHeader.aSamplingFactors[2] = 17;
		oFrameHeader.aQuantizationTableSelector[0] = 0;
		oFrameHeader.aQuantizationTableSelector[1] = 1;
		oFrameHeader.aQuantizationTableSelector[2] = 1;
	}
	else {
		cerr << "暂未支持的输入通道数." << endl;
		return;
	}

	/*--------------------扫描头初始化--------------------*/
	memset(&oScanHeader, 0, sizeof(ScanHeader));
	oScanHeader.nComponents = oFrameHeader.nComponents;
	oScanHeader.nSs = 0;
	oScanHeader.nSe = 63;
	oScanHeader.nA = 0;
	if (oScanHeader.nComponents == 1)
	{
		//TODO 支持直接的单通道处理,单通道需要检测是否正确初始化
		// oScanHeader.aComponentSelector[0] = 1;
		// oScanHeader.aHuffmanTablesSelector[0] = 0;

		oScanHeader.aComponentSelector[0] = 1;
		oScanHeader.aComponentSelector[1] = 2;
		oScanHeader.aComponentSelector[2] = 3;
		oScanHeader.aHuffmanTablesSelector[0] = 0;
		oScanHeader.aHuffmanTablesSelector[1] = 17;
		oScanHeader.aHuffmanTablesSelector[2] = 17;
	}
	else if (oScanHeader.nComponents == 3)
	{
		oScanHeader.aComponentSelector[0] = 1;
		oScanHeader.aComponentSelector[1] = 2;
		oScanHeader.aComponentSelector[2] = 3;
		oScanHeader.aHuffmanTablesSelector[0] = 0;
		oScanHeader.aHuffmanTablesSelector[1] = 17;
		oScanHeader.aHuffmanTablesSelector[2] = 17;
	}
	else {
		cerr << "暂未支持的输入通道数." << endl;
		return;
	}

	/*--------------------量化表初始化--------------------*/
	memset(aQuantizationTables, 0, 4 * sizeof(QuantizationTable));

	//根据编码质量修改量化表
	setQTByQuality(quality);

	//填充2张标准量化表(50%质量分数)
	aQuantizationTables[0].nPrecisionAndIdentifier = 0;
	memcpy(aQuantizationTables[0].aTable,
		getStaticTable(StaticTable::STD_Y_QT),
		getStaticTableLenth(StaticTable::STD_Y_QT));
	aQuantizationTables[1].nPrecisionAndIdentifier = 1;
	memcpy(aQuantizationTables[1].aTable,
		getStaticTable(StaticTable::STD_UV_QT),
		getStaticTableLenth(StaticTable::STD_UV_QT));

	//分配量化表设备缓冲区
	NPP_CHECK_CUDA(hipMalloc(&pdQuantizationTables, 64 * 4));

	////复制量化表到设备,因标准量化表已经是zigzag order,故而不需要Z形扫描,若量化表为顺序则需要
	//for (int i = 0; i < 4; ++i)
 //   {
 //       Npp8u temp[64];

 //       for (int k = 0; k < 32; ++k)
 //       {
 //           temp[2 * k + 0] = aQuantizationTables[i].aTable[getStaticTable(StaticTable::ZIGZAG)[k + 0]];
 //           temp[2 * k + 1] = aQuantizationTables[i].aTable[getStaticTable(StaticTable::ZIGZAG)[k + 32]];
 //       }

 //       NPP_CHECK_CUDA(hipMemcpyAsync(pdQuantizationTables + i * 64, temp, 64, hipMemcpyHostToDevice));
	//}

	NPP_CHECK_CUDA(hipMemcpyAsync(pdQuantizationTables,
		aQuantizationTables[0].aTable,
		getStaticTableLenth(StaticTable::STD_Y_QT),
		hipMemcpyHostToDevice));
	NPP_CHECK_CUDA(hipMemcpyAsync(pdQuantizationTables + getStaticTableLenth(StaticTable::STD_UV_QT),
		aQuantizationTables[1].aTable,
		getStaticTableLenth(StaticTable::STD_UV_QT),
		hipMemcpyHostToDevice));

	/*--------------------霍夫曼表初始化--------------------*/
	memset(aHuffmanTables, 0, 4 * sizeof(HuffmanTable));

	//初始化表指针
	pHuffmanDCTables = aHuffmanTables;//指向霍夫曼DC表
	pHuffmanACTables = &aHuffmanTables[2];//指向霍夫曼AC表

	//填充Huffman表
	aHuffmanTables[0].nClassAndIdentifier = 0;
	memcpy(aHuffmanTables[0].aCodes, getStaticTable(StaticTable::STD_DC_Y_NRCODES), getStaticTableLenth(StaticTable::STD_DC_Y_NRCODES));
	memcpy(aHuffmanTables[0].aTable, getStaticTable(StaticTable::STD_DC_Y_VALUES), getStaticTableLenth(StaticTable::STD_DC_Y_VALUES));

	aHuffmanTables[1].nClassAndIdentifier = 1;
	memcpy(aHuffmanTables[1].aCodes, getStaticTable(StaticTable::STD_DC_UV_NRCODES), getStaticTableLenth(StaticTable::STD_DC_UV_NRCODES));
	memcpy(aHuffmanTables[1].aTable, getStaticTable(StaticTable::STD_DC_UV_VALUES), getStaticTableLenth(StaticTable::STD_DC_UV_VALUES));

	aHuffmanTables[2].nClassAndIdentifier = 16;
	memcpy(aHuffmanTables[2].aCodes, getStaticTable(StaticTable::STD_AC_Y_NRCODES), getStaticTableLenth(StaticTable::STD_AC_Y_NRCODES));
	memcpy(aHuffmanTables[2].aTable, getStaticTable(StaticTable::STD_AC_Y_VALUES), getStaticTableLenth(StaticTable::STD_AC_Y_VALUES));

	aHuffmanTables[3].nClassAndIdentifier = 17;
	memcpy(aHuffmanTables[3].aCodes, getStaticTable(StaticTable::STD_AC_UV_NRCODES), getStaticTableLenth(StaticTable::STD_AC_UV_NRCODES));
	memcpy(aHuffmanTables[3].aTable, getStaticTable(StaticTable::STD_AC_UV_VALUES), getStaticTableLenth(StaticTable::STD_AC_UV_VALUES));

	//初始化霍夫曼编码表设备缓冲区,oScanHeader.aHuffmanTablesSelector[i]:(0,17,17)->(0,1,1)
	for (int i = 0; i < oFrameHeader.nComponents; ++i)
	{
		NPP_CHECK_NPP(nppiEncodeHuffmanSpecInitAlloc_JPEG(
			pHuffmanDCTables[(oScanHeader.aHuffmanTablesSelector[i] >> 4)].aCodes,
			NppiHuffmanTableType::nppiDCTable,
			&apHuffmanDCTableEncode[i]));

		NPP_CHECK_NPP(nppiEncodeHuffmanSpecInitAlloc_JPEG(
			pHuffmanACTables[(oScanHeader.aHuffmanTablesSelector[i] & 0x0f)].aCodes,
			NppiHuffmanTableType::nppiACTable,
			&apHuffmanACTableEncode[i]));
	}

	/*--------------------计算MCU分配图像显存--------------------*/
	//获取最大采样系数
	nMCUBlocksH = 0;
	nMCUBlocksV = 0;
	for (int i = 0; i < oFrameHeader.nComponents; ++i)
	{
		nMCUBlocksV = std::max(nMCUBlocksV, oFrameHeader.aSamplingFactors[i] & 0x0f);
		nMCUBlocksH = std::max(nMCUBlocksH, oFrameHeader.aSamplingFactors[i] >> 4);
	}

	//分配源图像设备定距内存
	for (int i = 0; i < oFrameHeader.nComponents; ++i)
	{
		NppiSize oBlocks;

		//最小编码单元(Minimum Coded Unit)中的数据单元的尺寸
		NppiSize oBlocksPerMCU = { oFrameHeader.aSamplingFactors[i] >> 4,
								   oFrameHeader.aSamplingFactors[i] & 0x0f };

		//图像宽高应该是MCU的整数倍
		oBlocks.width = (int)ceil((oFrameHeader.nWidth + 7) / 8 *
			static_cast<float>(oBlocksPerMCU.width) / nMCUBlocksH);
		oBlocks.width = DivUp(oBlocks.width, oBlocksPerMCU.width) * oBlocksPerMCU.width;

		oBlocks.height = (int)ceil((oFrameHeader.nHeight + 7) / 8 *
			static_cast<float>(oBlocksPerMCU.height) / nMCUBlocksV);
		oBlocks.height = DivUp(oBlocks.height, oBlocksPerMCU.height) * oBlocksPerMCU.height;

		aSrcSize[i].width = oBlocks.width * 8;
		aSrcSize[i].height = oBlocks.height * 8;

		//在设备上分配定距内存,apdDCT指定分配的设备指针,nPitch分配的间隔,请求分配的宽(apdDCT的类型Npp16s*8*8),高
		size_t nPitch;
		NPP_CHECK_CUDA(hipMallocPitch(&apdDCT[i], &nPitch, oBlocks.width * 64 * sizeof(Npp16s), oBlocks.height));
		aDCTStep[i] = static_cast<Npp32s>(nPitch);//图像宽度(以像素为单位)x 8 x sizeof(Npp16s)。

		//源图像定距设备缓冲区
		NPP_CHECK_CUDA(hipMallocPitch(&apdSrcImage[i], &nPitch, aSrcSize[i].width, aSrcSize[i].height));
		aSrcImageStep[i] = static_cast<Npp32s>(nPitch);

		//分配DCT锁页内存(解码时)
		//NPP_CHECK_CUDA(hipHostAlloc(&aphDCT[i], aDCTStep[i] * oBlocks.height, hipHostMallocDefault));
	}

	///计算基线霍夫曼编码的临时缓冲区的大小并分配编码缓冲区
	size_t nTempSize = 0;//临时设备缓冲区大小
	NPP_CHECK_NPP(nppiEncodeHuffmanGetSize(aSrcSize[0], oFrameHeader.nComponents,&nTempSize));
	NPP_CHECK_CUDA(hipMalloc(&pdJpegEncoderTemp, nTempSize));

#ifdef ONLY_IMAGE_COMPRESSION
	//合理分配扫描头大小并分配缓冲区
	Npp32s nScanSize = oFrameHeader.nWidth * oFrameHeader.nHeight * 2;//扫描头缓冲区长度
	nScanSize = nScanSize > (4 << 20) ? nScanSize : (4 << 20);//2048*1024
	NPP_CHECK_CUDA(hipMalloc(&pdScan, nScanSize));
#else 
	#ifdef ENABLE_IMAGE_SCALING
		//分配目标图像设备定距内存
		nScaleFactor = 1.0f;//TODO 缩放系数(提供接口,并且进一步支持缩放)
		oDstImageSize.width = (int)max(1.0f, floor((float)oFrameHeader.nWidth * nScaleFactor));
		oDstImageSize.height = (int)max(1.0f, floor((float)oFrameHeader.nHeight * nScaleFactor));
	#else 
		oDstImageSize.width = oFrameHeader.nWidth;
		oDstImageSize.height = oFrameHeader.nHeight;
	#endif // !ENABLE_IMAGE_SCALING

	for (int i = 0; i < oFrameHeader.nComponents; ++i) {

		NppiSize oBlocks;
		NppiSize oBlocksPerMCU = { oFrameHeader.aSamplingFactors[i] >> 4 ,
								   oFrameHeader.aSamplingFactors[i] & 0x0f };

		oBlocks.width = (int)ceil((oDstImageSize.width + 7) / 8 *
			static_cast<float>(oBlocksPerMCU.width) / nMCUBlocksH);
		oBlocks.width = DivUp(oBlocks.width, oBlocksPerMCU.width) * oBlocksPerMCU.width;

		oBlocks.height = (int)ceil((oDstImageSize.height + 7) / 8 *
			static_cast<float>(oBlocksPerMCU.height) / nMCUBlocksV);
		oBlocks.height = DivUp(oBlocks.height, oBlocksPerMCU.height) * oBlocksPerMCU.height;

		aDstSize[i].width = oBlocks.width * 8;
		aDstSize[i].height = oBlocks.height * 8;

		size_t nPitch;
		NPP_CHECK_CUDA(hipMallocPitch(&apDstImage[i], &nPitch, aDstSize[i].width, aDstSize[i].height));
		aDstImageStep[i] = static_cast<Npp32s>(nPitch);
	}

	//合理分配扫描头大小并分配缓冲区
	Npp32s nScanSize = oDstImageSize.width * oDstImageSize.height * 2;//扫描头缓冲区长度
	nScanSize = nScanSize > (4 << 20) ? nScanSize : (4 << 20);//2048*1024
	NPP_CHECK_CUDA(hipMalloc(&pdScan, nScanSize));
#endif // !ONLY_IMAGE_COMPRESSION

	/*--------------------写JPEG--------------------*/
	NPP_CHECK_CUDA(hipHostAlloc(&pDstJpeg, nScanSize, hipHostMallocDefault));

	nOutputLenth = 0;//写入缓冲区长度

	//TODO YUV格式判断后分配//PIX_FMT_YUVI420

	NPP_CHECK_CUDA(hipHostAlloc(&mY, oFrameHeader.nWidth * oFrameHeader.nHeight, hipHostMallocDefault));
	NPP_CHECK_CUDA(hipHostAlloc(&mU, oFrameHeader.nWidth / 2 * oFrameHeader.nHeight / 2, hipHostMallocDefault));//2:1采样,为Y的1/4大小
	NPP_CHECK_CUDA(hipHostAlloc(&mV, oFrameHeader.nWidth / 2 * oFrameHeader.nHeight / 2, hipHostMallocDefault));

	//uint32_t row_bytes = 4096 * 3;
	//NPP_CHECK_CUDA(hipMalloc(&mRGBData, row_bytes * 4096));
	
	/*--------------------异步流--------------------*/
	//分配及初始化多个stream,主要用于异步内存拷贝
	pStreams = (hipStream_t *)malloc(3 * sizeof(hipStream_t));

	for (int i = 0; i < 3; i++)
	{
		NPP_CHECK_CUDA(hipStreamCreate(&(pStreams[i])));
	}
}


CudaJpegEncoder::~CudaJpegEncoder()
{
	NPP_CHECK_NPP(nppiDCTFree(pDCTState));
	//NPP_CHECK_CUDA(hipHostFree(pSrcData));
	//NPP_CHECK_CUDA(hipFree(pdSrcData));

	NPP_CHECK_CUDA(hipFree(pdScan));
	NPP_CHECK_CUDA(hipFree(pdQuantizationTables));
	NPP_CHECK_CUDA(hipFree(pdJpegEncoderTemp));

	NPP_CHECK_CUDA(hipHostFree(mY));
	NPP_CHECK_CUDA(hipHostFree(mU));
	NPP_CHECK_CUDA(hipHostFree(mV));
	NPP_CHECK_CUDA(hipHostFree(pDstJpeg));

	for (int i = 0; i < oFrameHeader.nComponents; ++i)
	{
		NPP_CHECK_NPP(nppiEncodeHuffmanSpecFree_JPEG(apHuffmanDCTableEncode[i]));
		NPP_CHECK_NPP(nppiEncodeHuffmanSpecFree_JPEG(apHuffmanACTableEncode[i]));

		NPP_CHECK_CUDA(hipFree(apdDCT[i]));
		//NPP_CHECK_CUDA(hipHostFree(aphDCT[i]));

		NPP_CHECK_CUDA(hipFree(apdSrcImage[i]));
		NPP_CHECK_CUDA(hipFree(apdDstImage[i]));
	}

	for (int i = 0; i < 3; i++)
	{
		NPP_CHECK_CUDA(hipStreamDestroy(pStreams[i]));
	}

	//异常安全
	//free_s(pHuffmanDCTables);
	//free_s(pHuffmanACTables);
	free_s(pStreams);
	//释放new出的内存
	//deleteA_s(pSrcData);
}

//void CudaJpegEncoder::readImage(const char * file_name)
//{
//	// 打开文件流以读取
//	std::ifstream stream(file_name, std::ios::binary);
//
//	if (!stream.good())
//	{
//		return;
//	}
//	//获取流长度
//	stream.seekg(0, std::ios::end);//移动基地址至文件结束处，偏移为0
//	nInputLength = (int)stream.tellg();//获取指针的当前位置，即文件大小
//	stream.seekg(0, std::ios::beg);
//
//	//分配raw图像RGB数据内存
//	NPP_CHECK_CUDA(hipHostAlloc(pSrcData, nInputLength, hipHostMallocDefault));
//
//	//pSrcData = new unsigned char[nInputLength];
//	stream.read(reinterpret_cast<char *>(pSrcData), nInputLength);//读取
//}

//void CudaJpegEncoder::loadImage(unsigned char *pSrcData[3], int nInputLength)
//{
//	this->pSrcData[0] = pSrcData[0];
//	this->pSrcData[1] = pSrcData[1];
//	this->pSrcData[2] = pSrcData[2];
//	////分配raw图像RGB数据内存
//	//NPP_CHECK_CUDA(hipHostAlloc(&this->pSrcData, nInputLength, hipHostMallocDefault));
//
//	this->nInputLength = nInputLength;
//}

//void CudaJpegEncoder::UploadImage()
//{
//	//分配显存
//	NPP_CHECK_CUDA(hipMalloc(pdSrcData, nInputLength));
//	//上载图像
//	NPP_CHECK_CUDA(hipMemcpy(pdSrcData, pSrcData, nInputLength, hipMemcpyHostToDevice));
//}

//void CudaJpegEncoder::RGB2YUV()
//{
//
//	NppiSize oSizeROI = { aSrcSize->width,aSrcSize->height };
//	//RGB2YUV 平面2平面
//	//nppiRGBToYUV_8u_P3R(&pdSrcData, aSrcImageStep[0], apSrcImage, aDstImageStep[0], oSizeROI);
//	nppiRGBToYUV420_8u_P3R(pdSrcData, aSrcImageStep[0], apSrcImage, aDstImageStep, oSizeROI);
//	////RGB2YUV 压缩2平面
//	//nppiRGBToYUV_8u_C3P3R(pdSrcData, (int)aSrcImageStep[0], apSrcImage, (int)aSrcImageStep[0], oSizeROI);
//	//nppiRGBToYUV420_8u_C3P3R(pdSrcData, (int)aSrcImageStep[0], apSrcImage, aSrcImageStep, oSizeROI);
//}
//TODO 提供接收YUV图像及数据并处理的方法
void CudaJpegEncoder::setData(Npp8u * yuv_data, int yuv_fmt)
{
#ifdef DEBUG_MEASURE_KERNEL_TIME
	hipEvent_t start, stop;
	float elapsedTime;
	NPP_CHECK_CUDA(hipEventCreate(&start));
	NPP_CHECK_CUDA(hipEventCreate(&stop));
	NPP_CHECK_CUDA(hipEventRecord(start, 0));//0默认流
#endif // !DEBUG_MEASURE_KERNEL_TIME

	if (!yuv_data)
	{
		return;
	}
	uint32_t    off = 0;
	uint32_t    off_yuv = 0;
	uint32_t    half_h = oFrameHeader.nHeight >> 1;
	uint32_t    half_w = oFrameHeader.nWidth >> 1;

	for (int i = 0; i < oFrameHeader.nHeight; i++)
	{
		NPP_CHECK_CUDA(hipMemcpy(mY + off, yuv_data + off_yuv, oFrameHeader.nWidth, hipMemcpyHostToHost));//Y(HxW)复制到mY

		off += oFrameHeader.nWidth;
		off_yuv += oFrameHeader.nWidth;
	}

	switch (yuv_fmt)
	{
	case PixelFormat::PIX_FMT_YUVI420:
	{
		uint32_t uv_size = half_w * half_h;
		off_yuv = oFrameHeader.nWidth * oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			//从I420图像拆解UV,YYYYYYYYUUVV,单块U或V的大小为half_w*(half_h/2)
			//memcpy(mU + off, yuv_data + off_yuv, half_w);//跳过Y并把紧接着的俩个U交替复制到mU(分行存储,mU大小half_w*half_h)
			//memcpy(mV + off, yuv_data + off_yuv + uv_size, half_w);//跳过Y和U并把紧接着的俩个V交替复制到mV
			NPP_CHECK_CUDA(hipMemcpy(mU + off, yuv_data + off_yuv, half_w, hipMemcpyHostToHost));
			NPP_CHECK_CUDA(hipMemcpy(mV + off, yuv_data + off_yuv + uv_size, half_w, hipMemcpyHostToHost));
			off_yuv += half_w;
			off += half_w;
		}

		//for (int i = 0; i < half_h; i++)
		//{
		//	memset(mU + off, 128, half_w);//灰度图像Y=R=G=B,U=V=128
		//	memset(mV + off, 128, half_w);
		//	off_yuv += half_w;
		//	off += half_w;
		//}

		//if (mChannel == 3) {
		//	for (int i = 0; i < half_h; i++)
		//	{
		//		memcpy(mU + off, yuv_data + off_yuv, half_w);//从I420图像拆解UV
		//		memcpy(mV + off, yuv_data + off_yuv + half_size, half_w);
		//		off_yuv += half_w;
		//		off += half_w;
		//	}
		//}
		//else if (mChannel == 1) {
		//	for (int i = 0; i < half_h; i++)
		//	{
		//		memset(mU + off, 128, half_w);//灰度图像Y=R=G=B,U=V=128
		//		memset(mV + off, 128, half_w);
		//		off_yuv += half_w;
		//		off += half_w;
		//	}
		//}
		break;
	}
	case PixelFormat::PIX_FMT_NV12:
	{
		uint8_t*    yuv_ptr;
		uint8_t*    u_ptr;
		uint8_t*    v_ptr;
		off_yuv = oFrameHeader.nWidth * oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			yuv_ptr = yuv_data + off_yuv;
			u_ptr = mU + off;
			v_ptr = mV + off;
			for (int j = 0; j < oFrameHeader.nWidth; j += 2)
			{
				*u_ptr++ = *yuv_ptr++;//*u_ptr=*yuv_ptr;*u_ptr++;*yuv_ptr++;
				*v_ptr++ = *yuv_ptr++;//UVUV交替采样
			}
			off_yuv += oFrameHeader.nWidth;
			off += half_w;
		}
		break;
	}
	case PixelFormat::PIX_FMT_NV21:
	{
		uint8_t*    yuv_ptr;
		uint8_t*    u_ptr;
		uint8_t*    v_ptr;
		off_yuv = oFrameHeader.nWidth *  oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			yuv_ptr = yuv_data + off_yuv;
			u_ptr = mU + off;
			v_ptr = mV + off;
			for (int j = 0; j < oFrameHeader.nWidth; j += 2)
			{
				*v_ptr++ = *yuv_ptr++;//VUVU交替采样
				*u_ptr++ = *yuv_ptr++;
			}
			off_yuv += oFrameHeader.nWidth;
			off += half_w;
		}
		break;
	}
	default:
		cerr << "暂未支持的编码模式." << endl;
		break;
	}

	if (oFrameHeader.nComponents == 1) {
		//TODO 独立支持单通道图像编码
	}
	else if (oFrameHeader.nComponents == 3) {
		//定距内存对齐
		NPP_CHECK_CUDA(hipMemcpy2D(apdSrcImage[0],
			aSrcImageStep[0],
			mY,
			oFrameHeader.nWidth,
			oFrameHeader.nWidth,
			oFrameHeader.nHeight,
			hipMemcpyHostToDevice));
		NPP_CHECK_CUDA(hipMemcpy2D(apdSrcImage[1],
			aSrcImageStep[1],
			mU,
			oFrameHeader.nWidth / 2,
			oFrameHeader.nWidth / 2,
			oFrameHeader.nHeight / 2,
			hipMemcpyHostToDevice));
		NPP_CHECK_CUDA(hipMemcpy2D(apdSrcImage[2],
			aSrcImageStep[2],
			mV,
			oFrameHeader.nWidth / 2,
			oFrameHeader.nWidth / 2,
			oFrameHeader.nHeight / 2,
			hipMemcpyHostToDevice));
	}

#ifdef DEBUG_MEASURE_KERNEL_TIME

	NPP_CHECK_CUDA(hipEventRecord(stop, 0));
	NPP_CHECK_CUDA(hipEventSynchronize(stop));
	NPP_CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));
	printf_s("JPEG setData:0 (file:%s, line:%d) elapsed time : %f ms\n", __FILE__, __LINE__, elapsedTime);
	NPP_CHECK_CUDA(hipEventDestroy(start));
	NPP_CHECK_CUDA(hipEventDestroy(stop));
#endif // !DEBUG_MEASURE_KERNEL_TIME

}
//TODO 提供接收YUV图像及数据并处理的方法
void CudaJpegEncoder::setDataAsync(Npp8u * yuv_data, int yuv_fmt)
{
#ifdef DEBUG_MEASURE_KERNEL_TIME
	hipEvent_t start0, stop0;
	hipEvent_t start1, stop1;
	hipEvent_t start2, stop2;
	float elapsedTime0, elapsedTime1, elapsedTime2;
	NPP_CHECK_CUDA(hipEventCreate(&start0));
	NPP_CHECK_CUDA(hipEventCreate(&start1));
	NPP_CHECK_CUDA(hipEventCreate(&start2));
	NPP_CHECK_CUDA(hipEventCreate(&stop0));
	NPP_CHECK_CUDA(hipEventCreate(&stop1));
	NPP_CHECK_CUDA(hipEventCreate(&stop2));
	NPP_CHECK_CUDA(hipEventRecord(start0, pStreams[0]));
	NPP_CHECK_CUDA(hipEventRecord(start1, pStreams[1]));
	NPP_CHECK_CUDA(hipEventRecord(start2, pStreams[2]));
#endif // !DEBUG_MEASURE_KERNEL_TIME
	if (!yuv_data)
	{
		return;
	}
	uint32_t    off=0;
	uint32_t    off_yuv=0;
	uint32_t    half_h = oFrameHeader.nHeight >> 1;
	uint32_t    half_w = oFrameHeader.nWidth >> 1;

	for (int i = 0; i < oFrameHeader.nHeight; i++)
	{
		NPP_CHECK_CUDA(hipMemcpyAsync(mY + off, yuv_data + off_yuv, oFrameHeader.nWidth, hipMemcpyHostToHost,pStreams[0]));//Y(HxW)复制到mY

		off += oFrameHeader.nWidth;
		off_yuv += oFrameHeader.nWidth;
	}

	switch (yuv_fmt)
	{
	case PixelFormat::PIX_FMT_YUVI420:
	{
		uint32_t uv_size = half_w * half_h;
		off_yuv = oFrameHeader.nWidth * oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			//从I420图像拆解UV,YYYYYYYYUUVV,单块U或V的大小为half_w*(half_h/2)
			//memcpy(mU + off, yuv_data + off_yuv, half_w);//跳过Y并把紧接着的俩个U交替复制到mU(分行存储,mU大小half_w*half_h)
			//memcpy(mV + off, yuv_data + off_yuv + uv_size, half_w);//跳过Y和U并把紧接着的俩个V交替复制到mV
			NPP_CHECK_CUDA(hipMemcpyAsync(mU + off, yuv_data + off_yuv, half_w, hipMemcpyHostToHost, pStreams[1]));
			NPP_CHECK_CUDA(hipMemcpyAsync(mV + off, yuv_data + off_yuv + uv_size, half_w, hipMemcpyHostToHost, pStreams[2]));
			off_yuv += half_w;
			off += half_w;
		}

		//for (int i = 0; i < half_h; i++)
		//{
		//	memset(mU + off, 128, half_w);//灰度图像Y=R=G=B,U=V=128
		//	memset(mV + off, 128, half_w);
		//	off_yuv += half_w;
		//	off += half_w;
		//}

		//if (mChannel == 3) {
		//	for (int i = 0; i < half_h; i++)
		//	{
		//		memcpy(mU + off, yuv_data + off_yuv, half_w);//从I420图像拆解UV
		//		memcpy(mV + off, yuv_data + off_yuv + half_size, half_w);
		//		off_yuv += half_w;
		//		off += half_w;
		//	}
		//}
		//else if (mChannel == 1) {
		//	for (int i = 0; i < half_h; i++)
		//	{
		//		memset(mU + off, 128, half_w);//灰度图像Y=R=G=B,U=V=128
		//		memset(mV + off, 128, half_w);
		//		off_yuv += half_w;
		//		off += half_w;
		//	}
		//}
		break;
	}
	case PixelFormat::PIX_FMT_NV12:
	{
		uint8_t*    yuv_ptr;
		uint8_t*    u_ptr;
		uint8_t*    v_ptr;
		off_yuv = oFrameHeader.nWidth * oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			yuv_ptr = yuv_data + off_yuv;
			u_ptr = mU + off;
			v_ptr = mV + off;
			for (int j = 0; j < oFrameHeader.nWidth; j += 2)
			{
				*u_ptr++ = *yuv_ptr++;//*u_ptr=*yuv_ptr;*u_ptr++;*yuv_ptr++;
				*v_ptr++ = *yuv_ptr++;//UVUV交替采样
			}
			off_yuv += oFrameHeader.nWidth;
			off += half_w;
		}
		break;
	}
	case PixelFormat::PIX_FMT_NV21:
	{
		uint8_t*    yuv_ptr;
		uint8_t*    u_ptr;
		uint8_t*    v_ptr;
		off_yuv = oFrameHeader.nWidth *  oFrameHeader.nHeight;
		off = 0;

		for (int i = 0; i < half_h; i++)
		{
			yuv_ptr = yuv_data + off_yuv;
			u_ptr = mU + off;
			v_ptr = mV + off;
			for (int j = 0; j < oFrameHeader.nWidth; j += 2)
			{
				*v_ptr++ = *yuv_ptr++;//VUVU交替采样
				*u_ptr++ = *yuv_ptr++;
			}
			off_yuv += oFrameHeader.nWidth;
			off += half_w;
		}
		break;
	}
	default:
		cerr << "暂未支持的编码模式." << endl;
		break;
	}

	hipStreamSynchronize(pStreams[0]);
	hipStreamSynchronize(pStreams[1]);
	hipStreamSynchronize(pStreams[2]);

	if (oFrameHeader.nComponents == 1) {
		//TODO 独立支持单通道图像编码
	}
	else if (oFrameHeader.nComponents == 3) {
		//定距内存对齐
		NPP_CHECK_CUDA(hipMemcpy2DAsync(apdSrcImage[0],
									aSrcImageStep[0],
									mY,
									oFrameHeader.nWidth,
									oFrameHeader.nWidth,
									oFrameHeader.nHeight,
									hipMemcpyHostToDevice));
		NPP_CHECK_CUDA(hipMemcpy2DAsync(apdSrcImage[1],
									aSrcImageStep[1],
									mU,
									oFrameHeader.nWidth / 2,
									oFrameHeader.nWidth/2,
									oFrameHeader.nHeight / 2,
									hipMemcpyHostToDevice));
		NPP_CHECK_CUDA(hipMemcpy2DAsync(apdSrcImage[2],
									aSrcImageStep[2],
									mV,
									oFrameHeader.nWidth / 2,
									oFrameHeader.nWidth/2,
									oFrameHeader.nHeight / 2,
									hipMemcpyHostToDevice));
	}
	hipStreamSynchronize(pStreams[0]);
	hipStreamSynchronize(pStreams[1]);
	hipStreamSynchronize(pStreams[2]);

#ifdef DEBUG_MEASURE_KERNEL_TIME

	NPP_CHECK_CUDA(hipEventRecord(stop0, pStreams[0]));
	NPP_CHECK_CUDA(hipEventSynchronize(stop0));
	NPP_CHECK_CUDA(hipEventElapsedTime(&elapsedTime0, start0, stop0));
	printf_s("JPEG setData:pStreams[0] (file:%s, line:%d) elapsed time : %f ms\n", __FILE__, __LINE__, elapsedTime0);
	NPP_CHECK_CUDA(hipEventDestroy(start0));
	NPP_CHECK_CUDA(hipEventDestroy(stop0));

	NPP_CHECK_CUDA(hipEventRecord(stop1, pStreams[1]));
	NPP_CHECK_CUDA(hipEventSynchronize(stop1));
	NPP_CHECK_CUDA(hipEventElapsedTime(&elapsedTime1, start1, stop1));
	printf_s("JPEG setData:pStreams[1] (file:%s, line:%d) elapsed time : %f ms\n", __FILE__, __LINE__, elapsedTime1);
	NPP_CHECK_CUDA(hipEventDestroy(start1));
	NPP_CHECK_CUDA(hipEventDestroy(stop1));

	NPP_CHECK_CUDA(hipEventRecord(stop2, pStreams[2]));
	NPP_CHECK_CUDA(hipEventSynchronize(stop2));
	NPP_CHECK_CUDA(hipEventElapsedTime(&elapsedTime2, start2, stop2));
	printf_s("JPEG setData:pStreams[2] (file:%s, line:%d) elapsed time : %f ms\n", __FILE__, __LINE__, elapsedTime2);
	NPP_CHECK_CUDA(hipEventDestroy(start2));
	NPP_CHECK_CUDA(hipEventDestroy(stop2));
#endif // !DEBUG_MEASURE_KERNEL_TIME

}

void CudaJpegEncoder::EncodeJpeg()
{

#ifdef DEBUG_MEASURE_KERNEL_TIME
	hipEvent_t start, stop;
	float elapsedTime;
	NPP_CHECK_CUDA(hipEventCreate(&start));
	NPP_CHECK_CUDA(hipEventRecord(start, 0));//0默认流
#endif // !DEBUG_MEASURE_KERNEL_TIME

#ifdef ENABLE_IMAGE_SCALING
	/*------------------------------缩放支持------------------------------*/
	// 缩放到目标图像大小
	// 只处理420图像
	int aSampleFactor[3] = { 1, 2, 2 };
	for (int i = 0; i < mChannel; ++i)
	{
		NppiSize oBlocksPerMCU = { oFrameHeader.aSamplingFactors[i] >> 4, oFrameHeader.aSamplingFactors[i] & 0x0f };
		NppiSize oSrcImageSize = { (oFrameHeader.nWidth * oBlocksPerMCU.width) / nMCUBlocksH, (oFrameHeader.nHeight * oBlocksPerMCU.height) / nMCUBlocksV };
		NppiRect oSrcImageROI = { 0,0,oSrcImageSize.width, oSrcImageSize.height };
		NppiRect oDstImageROI;
		oDstImageROI.x = 0;
		oDstImageROI.y = 0;
		oDstImageROI.width = oDstImageSize.width / aSampleFactor[i];
		oDstImageROI.height = oDstImageSize.height / aSampleFactor[i];

		NppiInterpolationMode eInterploationMode = NPPI_INTER_SUPER;

		if (nScaleFactor >= 1.f)
			eInterploationMode = NPPI_INTER_LANCZOS;

		NPP_CHECK_NPP(nppiResize_8u_C1R(apSrcImage[i], aSrcImageStep[i], oSrcImageSize, oSrcImageROI,
			apDstImage[i], aDstImageStep[i], oDstImageSize, oDstImageROI, eInterploationMode));
	}
#else
	//重定向
	for (size_t i = 0; i < oFrameHeader.nComponents; i++)
	{
		apdDstImage[i] = apdSrcImage[i];
		aDstImageStep[i] = aSrcImageStep[i];
		aDstSize[i] = aSrcSize[i];

		apdSrcImage[i] = NULL;
		aSrcImageStep[i] = NULL;
	}
#endif // !ENABLE_IMAGE_SCALING



	/*------------------------------前向DCT及量化------------------------------*/
	for (int i = 0; i < oFrameHeader.nComponents; ++i)
	{
		NPP_CHECK_NPP(nppiDCTQuantFwd8x8LS_JPEG_8u16s_C1R_NEW(
			apdDstImage[i],
			aDstImageStep[i],
			apdDCT[i],
			aDCTStep[i],
			pdQuantizationTables + oFrameHeader.aQuantizationTableSelector[i] * 64,
			aDstSize[i],
			pDCTState));
	}

	/*------------------------------霍夫曼编码------------------------------*/
	if (oFrameHeader.nComponents == 1) {
		Npp8u * hpCodesDC[3];
		Npp8u * hpCodesAC[3];
		Npp8u * hpTableDC[3];
		Npp8u * hpTableAC[3];
		for (int i = 0; i < 2; ++i)
		{
			hpCodesDC[i] = pHuffmanDCTables[i].aCodes;
			hpCodesAC[i] = pHuffmanACTables[i].aCodes;
			hpTableDC[i] = pHuffmanDCTables[i].aTable;
			hpTableAC[i] = pHuffmanACTables[i].aTable;
		}
		//TODO 单通道霍夫曼编码
		NPP_CHECK_NPP(nppiEncodeOptimizeHuffmanScan_JPEG_8u16s_P1R(
			*apdDCT,
			aDCTStep[0],
			0,
			oScanHeader.nSs,
			oScanHeader.nSe,
			oScanHeader.nA >> 4,
			oScanHeader.nA & 0x0f,
			pdScan,
			&nScanLength,
			hpCodesDC[0],
			hpTableDC[0],
			hpCodesAC[0],
			hpTableAC[0],
			*apHuffmanDCTableEncode,
			*apHuffmanACTableEncode,
			aDstSize[0],
			pdJpegEncoderTemp));
	}
	else if (oFrameHeader.nComponents == 3) {
		Npp8u * hpCodesDC[3];
		Npp8u * hpCodesAC[3];
		Npp8u * hpTableDC[3];
		Npp8u * hpTableAC[3];
		for (int i = 0; i < 2; ++i)
		{
			hpCodesDC[i] = pHuffmanDCTables[i].aCodes;
			hpCodesAC[i] = pHuffmanACTables[i].aCodes;
			hpTableDC[i] = pHuffmanDCTables[i].aTable;
			hpTableAC[i] = pHuffmanACTables[i].aTable;
		}
		//霍夫曼3通道优化编码
		NPP_CHECK_NPP(nppiEncodeOptimizeHuffmanScan_JPEG_8u16s_P3R(
			apdDCT,
			aDCTStep,
			0,
			oScanHeader.nSs,
			oScanHeader.nSe,
			oScanHeader.nA >> 4,
			oScanHeader.nA & 0x0f,
			pdScan,
			&nScanLength,
			hpCodesDC,
			hpTableDC,
			hpCodesAC,
			hpTableAC,
			apHuffmanDCTableEncode,
			apHuffmanACTableEncode,
			aDstSize,
			pdJpegEncoderTemp));
		//优化非优化似乎没有区别
		 //NPP_CHECK_NPP(nppiEncodeHuffmanScan_JPEG_8u16s_P3R(
		 //	apdDCT,
		 //	aDCTStep,
		 //	0,
		 //	oScanHeader.nSs,
		 //	oScanHeader.nSe,
		 //	oScanHeader.nA >> 4,
		 //	oScanHeader.nA & 0x0f,
		 //	pdScan, &nScanLength,
		 //	apHuffmanDCTableEncode,
		 //	apHuffmanACTableEncode,
		 //	aDstSize,
		 //	pJpegEncoderTemp));
	}
	else {
		cerr << "暂未支持的输入通道数." << endl;
		return;
	}

	/*------------------------------写数据段------------------------------*/
	unsigned char *pDstOutput;//最终编码主机数据(写入缓冲区尾指针)
	pDstOutput = pDstJpeg;//指向头指针
	writeMarker(SOI, pDstOutput);//写SOI,图像开始
	writeJFIFTag(pDstOutput);//写APP0

	writeQuantizationTable(aQuantizationTables[0], pDstOutput);//写量化表
	writeQuantizationTable(aQuantizationTables[1], pDstOutput);

	writeFrameHeader(oFrameHeader, pDstOutput);//写图像头

	writeHuffmanTable(pHuffmanDCTables[0], pDstOutput);//写霍夫曼表
	writeHuffmanTable(pHuffmanACTables[0], pDstOutput);
	writeHuffmanTable(pHuffmanDCTables[1], pDstOutput);
	writeHuffmanTable(pHuffmanACTables[1], pDstOutput);

	writeScanHeader(oScanHeader, pDstOutput);//写扫描头
	NPP_CHECK_CUDA(hipMemcpy(pDstOutput, pdScan, nScanLength, hipMemcpyDeviceToHost));//设备扫描头缓冲区数据复制回主机
	pDstOutput += nScanLength;
	writeMarker(EOI, pDstOutput);//图像结束

	nOutputLenth = static_cast<int>(pDstOutput - pDstJpeg);

#ifdef DEBUG_MEASURE_KERNEL_TIME
	NPP_CHECK_CUDA(hipEventCreate(&stop));
	NPP_CHECK_CUDA(hipEventRecord(stop, 0));
	NPP_CHECK_CUDA(hipEventSynchronize(stop));
	NPP_CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));
	printf_s("JPEG encode: (file:%s, line:%d) elapsed time : %f ms\n", __FILE__, __LINE__, elapsedTime);
	NPP_CHECK_CUDA(hipEventDestroy(start));
	NPP_CHECK_CUDA(hipEventDestroy(stop));
#endif // !DEBUG_MEASURE_KERNEL_TIME
}

